
#include <iostream>
using namespace std;
#include <hip/hip_runtime_api.h>

void gpu_test()
{
  cout << endl;

  int deviceCount = -1;
  checkCudaErrors(hipGetDeviceCount(&deviceCount));
  if (deviceCount == 0) {
    fprintf(stderr, "error: no devices supporting CUDA.\n");
    exit(EXIT_FAILURE);
  }

  cout << " deviceCount " << deviceCount  << endl;
  
  int dev = 0;
  hipDeviceProp_t prop;
  checkCudaErrors(hipGetDeviceProperties(&prop, dev));
  cout << " Using device: " << dev << "\n"
       << " Name: " << prop.name << ","
       << " Global mem: " << prop.totalGlobalMem/1024.0/1024/1024 << ","
       << " Compute v" << (int) prop.major << "." << (int)prop.minor << ","
       << " Clock: " << (int) prop.clockRate << " KHz" << endl;
  
  cout << endl;
}
